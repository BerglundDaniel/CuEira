#ifndef CUDAADAPTER_H
#define CUDAADAPTER_H

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <string>

#include <CudaException.h>
#include <CublasException.h>
#include <DimensionMismatch.h>

namespace CuEira {
namespace CUDA {

/**
 * These are some functions to wrap Cuda
 *
 * @author Daniel Berglund daniel.k.berglund@gmail.com
 */

/**
 * Throws CudaException if error is not hipSuccess with the message as the string for the exception and the string for the Cuda error.
 */
inline void handleCudaStatus(hipError_t error, std::string message) {
  if(error != hipSuccess){
    message.append(hipGetErrorString(error));
    throw CudaException(message.c_str());
  }
}

/**
 * Allocate memory for the PRECISION pointer with size number * sizeof(PRECISION) on the GPU, throws CudaException if there is an error
 */
inline void allocateDeviceMemory(void** pointerDevice, int number) {
  handleCudaStatus(hipMalloc(pointerDevice, number * sizeof(PRECISION)), "Device memory allocation failed: ");
}

/**
 * Allocate pinned memory for the PRECISION pointer with size number * sizeof(PRECISION), throws CudaException if there is an error
 */
inline void allocateHostPinnedMemory(void** pointerDevice, int number) {
  handleCudaStatus(hipHostAlloc(pointerDevice, number * sizeof(PRECISION), hipHostMallocPortable),
      "Host pinned memory allocation failed: ");
}

/**
 * Free memory on the GPU, throws CudaException if there is an error
 */
inline void freeDeviceMemory(void* pointerDevice) {
  handleCudaStatus(hipFree(pointerDevice), "Freeing device memory failed: ");
}

/**
 * Free pinned memory, throws CudaException if there is an error
 */
inline void freePinnedMemory(void* pointerDevice) {
  handleCudaStatus(hipHostFree(pointerDevice), "Freeing host memory failed: ");
}

} /* namespace CUDA */
} /* namespace CuEira */

#endif // CUDAADAPTER_H
