#ifndef CUDAADAPTER_H
#define CUDAADAPTER_H

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <string>

#include <CudaException.h>
#include <CublasException.h>
#include <DimensionMismatch.h>

namespace CuEira {
namespace CUDA {

/**
 * These are some functions to wrap Cuda
 *
 * @author Daniel Berglund daniel.k.berglund@gmail.com
 */

/**
 * Convert Cublas error enum to string
 */
inline static const std::string cublasGetErrorString(hipblasStatus_t error) {
  switch(error){
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";

  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";

  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";

  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";

  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";

  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";

  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";

  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }

  return "<unknown>";
}

/**
 * Throws CudaException if error is not hipSuccess with the message as the string for the exception and the string for the Cuda error.
 */
inline void handleCudaStatus(hipError_t error, std::string message) {
  if(error != hipSuccess){
    message.append(hipGetErrorString(error));
    throw CudaException(message.c_str());
  }
}

/**
 * Throws CudaException if status is not HIPBLAS_STATUS_SUCCESS with the message as the string for the exception and the string for the Cuda error.
 */
inline void handleCublasStatus(hipblasStatus_t status, std::string message) {
  if(status != HIPBLAS_STATUS_SUCCESS){
    message.append(cublasGetErrorString(status)); //FIXME
    throw CublasException(message.c_str());
  }
}

/**
 * Allocate memory for the PRECISION pointer with size number * sizeof(PRECISION) on the GPU, throws CudaException if there is an error
 */
inline void allocateDeviceMemory(void** pointerDevice, int number) {
  handleCudaStatus(hipMalloc(pointerDevice, number * sizeof(PRECISION)), "Device memory allocation failed: ");
}

/**
 * Allocate pinned memory for the PRECISION pointer with size number * sizeof(PRECISION), throws CudaException if there is an error
 */
inline void allocateHostPinnedMemory(void** pointerDevice, int number) {
  handleCudaStatus(hipHostAlloc(pointerDevice, number * sizeof(PRECISION), hipHostMallocPortable),"Pinned device memory allocation failed: ");
}

/**
 * Free memory on the GPU, throws CudaException if there is an error
 */
inline void freeDeviceMemory(void* pointerDevice) {
  handleCudaStatus(hipFree(pointerDevice), "Freeing device memory failed: ");
}

/**
 * Free pinned memory, throws CudaException if there is an error
 */
inline void freePinnedMemory(void* pointerDevice) {
  handleCudaStatus(hipHostFree(pointerDevice), "Freeing host memory failed: ");
}

} /* namespace CUDA */
} /* namespace CuEira */

#endif // CUDAADAPTER_H
