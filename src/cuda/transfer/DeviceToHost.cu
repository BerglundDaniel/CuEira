#include "DeviceToHost.h"

namespace CuEira {
namespace CUDA {

DeviceToHost::DeviceToHost(const hipStream_t& cudaStream) :
    cudaStream(cudaStream) {

}

DeviceToHost::~DeviceToHost() {

}

HostMatrix* DeviceToHost::transferMatrix(const DeviceMatrix* matrixDevice) const {
  const int numberOfRows = matrixDevice->getNumberOfRows();
  const int numberOfColumns = matrixDevice->getNumberOfColumns();

  PinnedHostMatrix* hostMatrix = new PinnedHostMatrix(numberOfRows, numberOfColumns);
  PRECISION* matrixPointerHost = hostMatrix->getMemoryPointer();
  const PRECISION* matrixPointerDevice = matrixDevice->getMemoryPointer();

  handleCublasStatus(
      hipblasGetMatrixAsync(numberOfRows, numberOfColumns, sizeof(PRECISION), matrixPointerDevice, numberOfRows,
          matrixPointerHost, numberOfRows, cudaStream), "Error when transferring matrix from device to host: ");

  return hostMatrix;
}

HostVector* DeviceToHost::transferVector(const DeviceVector* vectorDevice) const {
  const int numberOfRows = vectorDevice->getNumberOfRows();

  PinnedHostVector* hostVector = new PinnedHostVector(numberOfRows);
  PRECISION* vectorPointerHost = hostVector->getMemoryPointer();
  const PRECISION* vectorPointerDevice = vectorDevice->getMemoryPointer();

  handleCublasStatus(
      hipblasGetVectorAsync(numberOfRows, sizeof(PRECISION), vectorPointerDevice, 1, vectorPointerHost, 1, cudaStream),
      "Error when transferring vector from device to host point: ");

  return hostVector;
}

void DeviceToHost::transferMatrix(const DeviceMatrix* matrixDevice, PRECISION* hostMemoryPosition) const {
  const int numberOfRows = matrixDevice->getNumberOfRows();
  const int numberOfColumns = matrixDevice->getNumberOfColumns();

  const PRECISION* matrixPointerDevice = matrixDevice->getMemoryPointer();

  handleCublasStatus(
      hipblasGetMatrixAsync(numberOfRows, numberOfColumns, sizeof(PRECISION), matrixPointerDevice, numberOfRows,
          hostMemoryPosition, numberOfRows, cudaStream), "Error when transferring matrix from device to host: ");

}

void DeviceToHost::transferVector(const DeviceVector* vectorDevice, PRECISION* hostMemoryPosition) const {
  const int numberOfRows = vectorDevice->getNumberOfRows();

  const PRECISION* vectorPointerDevice = vectorDevice->getMemoryPointer();

  handleCublasStatus(
      hipblasGetVectorAsync(numberOfRows, sizeof(PRECISION), vectorPointerDevice, 1, hostMemoryPosition, 1, cudaStream),
      "Error when transferring vector from device to host point: ");
}

} /* namespace CUDA */
} /* namespace CuEira */
