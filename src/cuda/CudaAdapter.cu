#include "CudaAdapter.h"

namespace CuEira {
namespace CUDA {

void handleCudaStatus(hipError_t error, std::string message) {
  if(error != hipSuccess){
    message.append(hipGetErrorString(error));
    throw CudaException(message.c_str());
  }
}

void allocateDeviceMemory(void** pointerDevice, int number) {
  handleCudaStatus(hipMalloc(pointerDevice, number * sizeof(PRECISION)), "Device memory allocation failed: ");
}

void allocateHostPinnedMemory(void** pointerDevice, int number) {
  handleCudaStatus(hipHostAlloc(pointerDevice, number * sizeof(PRECISION), hipHostMallocPortable),
      "Host pinned memory allocation failed: ");
}

void freeDeviceMemory(void* pointerDevice) {
  handleCudaStatus(hipFree(pointerDevice), "Freeing device memory failed: ");
}

void freePinnedMemory(void* pointerDevice) {
  handleCudaStatus(hipHostFree(pointerDevice), "Freeing host memory failed: ");
}

} /* namespace CUDA */
} /* namespace CuEira */
