#include "Device.h"

namespace CuEira {
namespace CUDA {

Device::Device(int deviceNumber, const Container::DeviceVector* outcomes) :
    deviceNumber(deviceNumber), outcomes(outcomes) {

}

Device::~Device() {
  delete outcomes;
}

bool Device::isActive() const {
  int* activeDeviceNumber = new int(-1);
  hipGetDevice(activeDeviceNumber);

  if(*activeDeviceNumber == deviceNumber){
    return true;
  }else{
    return false;
  }
}

bool Device::setActiveDevice() const {
  hipError_t status = hipSetDevice(deviceNumber);

  if(status == hipSuccess){
    return true;
  }else{
    return false;
  }
}

const Container::DeviceVector& Device::getOutcomes() const {
  return *outcomes;
}

} /* namespace CUDA */
} /* namespace CuEira */
